#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <SDL.h>
#include <SDL_image.h>
#include <hip/hip_complex.h>

#define LIMIT 4
#define IMG_SIZE 4096
#define MAX_ITERATIONS 250

__device__ static Uint32 hslToRGB(int angle);

__device__ static void setPixelRGB(Uint32 *pixels, const int x, const int y,
	       		       	   const int r, const int g, const int b) {
    pixels[y * IMG_SIZE + x] = (0xff << 24) | (r << 16) | (g << 8) | b;
}

__device__ static void setPixelHSL(Uint32 *pixels, const int x, const int y, const int angle) {
	pixels[y * IMG_SIZE + x] = hslToRGB(angle);
}

// MAGIC!! From: http://stackoverflow.com/questions/2353211/hsl-to-rgb-color-conversion
__device__ static double hueToRGB(const double p, const double q, double t) {
	if (t < 0.0)
		t += 1.0;
	
	if (t > 1)
		t -= 1.0;

	if (t < (1.0/6.0))
		return p + (q - p) * 6.0 * t;

	if (t < 0.5)
		return q;

	if (t < (2.0/3.0))
		return p + (q - p) * ((2.0/3.0) - t) * 6.0;

	return p;
}

__device__ static Uint32 hslToRGB(const int angle) {
	double r, g, b;
	double h = (angle % 360) / 360.0;
	//printf("Hue: %f\n", h);

	double q = 1.0; 
	double p = 0.0;
	r = hueToRGB(p, q, h + 1.0/3.0);
	g = hueToRGB(p, q, h);
	b = hueToRGB(p, q, h - 1.0/3.0);

	//printf("RGB: %f, %f, %f\n", r, g, b);

	int red = (int) round(r * 255.0);
	int green = (int) round(g * 255.0);
	int blue = (int) round(b * 255.0);

	return (0xff << 24) | (red << 16) | (green << 8) | blue;
}

__global__ static void mandelbrot(Uint32 *image, const double xmin, const double xmax,
                                  const double ymin, const double ymax) {
    const double xstep = (xmax - xmin) / IMG_SIZE;
    const double ystep = (ymax - ymin) / IMG_SIZE;

    const int xpix = blockIdx.x * blockDim.x + threadIdx.x;
    const int ypix = blockIdx.y;

    hipDoubleComplex c = make_hipDoubleComplex(xmin + xpix * xstep, ymin + ypix * ystep);
    hipDoubleComplex z = make_hipDoubleComplex(0.0, 0.0);

    int count = 0;
    
    while (count < MAX_ITERATIONS && hipCabs(z) < LIMIT) {
        count++;

        z = hipCmul(z, z);
        z = hipCadd(z, c);
    }

    if (count >= MAX_ITERATIONS) {
        setPixelRGB(image, xpix, ypix, 0, 0, 0);
    } else {
        setPixelHSL(image, xpix, ypix, count);
    }
}

int main(int argc, char *argv[]) {
    if (SDL_Init(0) != 0) {
        SDL_Log("Unable to initialize SDL: %s\n", SDL_GetError());
        return 1;
    }

    if (IMG_Init(IMG_INIT_PNG) != IMG_INIT_PNG) {
        SDL_Log("Unable to initialize SDL_Image: %s\n", IMG_GetError());
        return 2;
    }

    SDL_Surface *image = IMG_Load("out.png");
    if (!image) {
        SDL_Log("IMG_Load error: %s\n", IMG_GetError());
	    return 3;
    }

    printf("Height: %d, Width: %d\n", image->h, image->w);
    if (image->h != IMG_SIZE && image->w != IMG_SIZE) {
	    fprintf(stderr, "Unexpected image dimensions: (%d, %d). Expected (%d, %d).\n", image->w, image->h, IMG_SIZE, IMG_SIZE);
	    return 4;
    }

    Uint32 *pixels;
    hipMallocManaged(&pixels, IMG_SIZE * IMG_SIZE * sizeof(Uint32));
    delete [] (Uint32 *)image->pixels;
    image->pixels = pixels;

    const double xmin = -0.7463;
    const double xmax = -0.7473;
    const double ymin = 0.1102;
    const double ymax = 0.1112;
    /*const double xmin = -2;
    const double xmax = 1;
    const double ymin = -1.5;
    const double ymax = 1.5;*/

    int threadsPerBlock = 256;
    int numBlocks = IMG_SIZE / threadsPerBlock;
    dim3 grid(numBlocks, IMG_SIZE);
    mandelbrot<<<grid, threadsPerBlock>>>(pixels, xmin, xmax, ymin, ymax);
    hipDeviceSynchronize();

//    (Uint32 *) (image->pixels) = pixels;
    //Uint32 *imgPixels = (Uint32 *) image->pixels;
    //for (int i = 0; i < IMG_SIZE * IMG_SIZE; i++) {
//        imgPixels[i] = pixels[i];
 //   }

    /*
    int size = 4;
    int tPB = 2;
    int numBlocks = size / tPB;
    dim3 grid(numBlocks, size);
    test<<<grid, tPB>>>();
    hipDeviceSynchronize();
    */


    //mandelbrot(image);

    //Uint32 pixelformat = image->format->format;
    //const char *formatName = SDL_GetPixelFormatName(pixelformat);
    //printf("Format: %s\n", formatName);
    //printf("BytesPerPixel: %d\n", image->format->BytesPerPixel);
    IMG_SavePNG(image, "out.png");
    
    hipFree(pixels);
    //SDL_FreeSurface(image); Causes segfault
    IMG_Quit();
    SDL_Quit();

    return 0;
}
